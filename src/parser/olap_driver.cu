#include "hip/hip_runtime.h"
#include <cctype>
#include <fstream>
#include <cassert>
#include <fcntl.h>
#include <sys/types.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/mman.h>

//GPU libs
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>

#include "olap_driver.hh"
#include "olap_parser.hh"
#include "olap_scanner.hh"

struct is_newline_break{                                                                               
  __host__ __device__                                                           
    bool operator()(const char x)                                               
    {                                                                           
      return x == 10;                                                           
    }                                                                           
};

struct parse_functor
{
  const char *source;
  char **dest;
  const unsigned int *ind;
  const unsigned int *cnt;
  const char *separator;
  const int *src_ind;
  const unsigned int *dest_len;

  parse_functor(
      const char* _source, char** _dest, 
      const unsigned int* _ind, const unsigned int* _cnt, 
      const char* _separator,
      const int* _src_ind, const unsigned int* _dest_len
      ):
    source(_source), dest(_dest), ind(_ind), cnt(_cnt),  separator(_separator), src_ind(_src_ind), dest_len(_dest_len) {}

  template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
      unsigned int curr_cnt = 0, dest_curr = 0, j = 0, t, pos;
      pos = src_ind[i]+1;

      while(dest_curr < *cnt) {
        if(ind[dest_curr] == curr_cnt) { //process
          t = 0;
          while(source[pos+j] != *separator) {
            if(source[pos+j] != 0) {
              dest[dest_curr][dest_len[dest_curr]*i+t] = source[pos+j];
              t++;
            };
            j++;
          };
          j++;
          dest_curr++;
        }
        else {
          while(source[pos+j] != *separator) {
            j++;
          };
          j++;
        };
        curr_cnt++;
      }
    }
};

namespace OLAP{
  OLAP_Driver::~OLAP_Driver(){
    delete(scanner);
    scanner = nullptr;
    delete(parser);
    parser = nullptr;
  }

  void OLAP_Driver::parse( const char * const filename ){
    assert( filename != nullptr );
    std::ifstream in_file( filename );
    if( ! in_file.good() ){
      exit( EXIT_FAILURE );
    }
    parse_helper( in_file );
    return;
  }

  void OLAP_Driver::parse( std::istream &stream ){
    if( ! stream.good()  && stream.eof() ){
      return;
    }
    //else
    parse_helper( stream );
    return;
  }


  void OLAP_Driver::parse_helper( std::istream &stream ){
    delete(scanner);
    try{
      scanner = new OLAP::OLAP_Scanner( &stream );
    }
    catch( std::bad_alloc &ba ){
      std::cerr << "Failed to allocate scanner: (" <<
        ba.what() << "), exiting!!\n";
      exit( EXIT_FAILURE );
    }

    delete(parser);
    try{
      parser = new OLAP::OLAP_Parser( (*scanner) /* scanner */,
          (*this) /* driver */ );
    }
    catch( std::bad_alloc &ba ){
      std::cerr << "Failed to allocate parser: (" <<
        ba.what() << "), exiting!!\n";
      exit( EXIT_FAILURE );
    }
    const int accept( 0 );
    if( parser->parse() != accept ){
      std::cerr << "Parse failed!!\n";
    }
    return;
  }

  void OLAP_Driver::load_matrix_csc ( std::string  filename, int col_number, int max_col_size ){
    std::cout << "loading column " << col_number << " from file " << filename << std::endl;
    std::clock_t start1 = std::clock();
    int fd;
    fd = open ( filename.c_str(), O_RDONLY );
    if ( fd == -1 ){
      perror("open");
    }

    off_t file_size;
    file_size = lseek(fd, 0, SEEK_END);
    thrust::device_vector<char> dev(file_size);
    char* mapped_file;

    mapped_file = (char*)mmap (0, file_size, PROT_READ, MAP_SHARED, fd, 0);                  

    if (mapped_file == MAP_FAILED){                                                        
      perror ("mmap");                                                            
    }                                                                             

    if (close (fd) == -1){                                                       
      perror ("close");                                                           
    }                                                                             

    thrust::copy(mapped_file, mapped_file+file_size, dev.begin());                                     
    std::cout << "going to count" << std::endl;
    int line_count = std::count(dev.begin(), dev.end(), '\n');                        
    std::cout << "There are " << line_count << " total lines in a file with size " << file_size << std::endl;    

    // find out the position of every newline
    thrust::device_vector<int> dev_newline_pos(line_count+1); 
    thrust::copy_if(
        thrust::make_counting_iterator((unsigned int) 0 ), 
        thrust::make_counting_iterator((unsigned int) file_size), 
        dev.begin(), dev_newline_pos.begin()+1, 
        is_newline_break()
        ); 
    for ( int pos = 0; pos < dev_newline_pos.size(); pos++ ){
      std::cout << dev_newline_pos[pos] << std::endl;
    }
    // field position based on column number
    thrust::device_vector<unsigned int> field_index(1);
    field_index[0]=col_number;

    // field separator
    thrust::device_vector<char> field_separator(1);
    field_separator[0] = '|';

    thrust::device_vector<char> dev_res1(line_count*max_col_size); 
    thrust::fill(dev_res1.begin(), dev_res1.end(), 0); 

    thrust::device_vector<char*> dest(1);
    dest[0] = thrust::raw_pointer_cast(dev_res1.data());

    //fields max lengths 
    thrust::device_vector<unsigned int> dest_len(1); 
    dest_len[0] = max_col_size;

    //fields count
    thrust::device_vector<unsigned int> ind_cnt(1); 
    ind_cnt[0] = 1;

    thrust::counting_iterator<unsigned int> begin(0);

    parse_functor ff(
        (const char*) thrust::raw_pointer_cast(dev.data()),
        (char**) thrust::raw_pointer_cast(dest.data()), 
        thrust::raw_pointer_cast(field_index.data()),
        thrust::raw_pointer_cast(ind_cnt.data()), 
        thrust::raw_pointer_cast(field_separator.data()), 
        thrust::raw_pointer_cast(dev_newline_pos.data()), 
        thrust::raw_pointer_cast(dest_len.data())
        );
    thrust::for_each(begin, begin + line_count, ff);
    
    for (int pos = 0; pos < (line_count * max_col_size ); pos ++  ){
      std::cout << dev_res1[pos];
      if( (pos % (max_col_size+1) )== 0 ){
        std::cout << std::endl;
      }
    }
    std::cout << "leaving load matrix " << std::endl;
  }

  std::ostream& OLAP_Driver::print( std::ostream &stream ){
    stream << red  << "Debug info: " << norm << "\n";
    stream << blue << "OLAP: "  << norm << "\n";

    return(stream);
  }
}


