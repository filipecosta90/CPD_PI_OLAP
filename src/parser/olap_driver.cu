#include "hip/hip_runtime.h"
#include <cctype>
#include <fstream>
#include <cassert>
#include <fcntl.h>
#include <sys/types.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/mman.h>

//GPU libs
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>

#include "olap_driver.hh"
#include "olap_parser.hh"
#include "olap_scanner.hh"

struct is_newline_break{                                                                               
  __host__ __device__                                                           
    bool operator()(const char x)                                               
    {                                                                           
      return x == '\n';                                                           
    }                                                                           
};

struct parse_functor
{
  const char *source;
  int **dest_col_start;
  int **dest_col_size;
  const unsigned int *ind;
  const unsigned int *num_cols_to_parse;
  const char *field_separator;
  const int *src_newline_ind;
  const unsigned int *dest_len;

  parse_functor(
      const char* _source, int** _dest_col_start, int** _dest_col_size, 
      const unsigned int* _ind, const unsigned int* _cnt, 
      const char* _separator,
      const int* _src_ind, const unsigned int* _dest_len
      ):
    source(_source), dest_col_start(_dest_col_start), dest_col_size(_dest_col_size), ind(_ind), num_cols_to_parse(_cnt),  field_separator(_separator), src_newline_ind(_src_ind), dest_len(_dest_len) {}

  template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
      unsigned int current_column = 0, total_parsed_cols = 0, inline_pos = 0, in_col_nonzeros, line_start;
      line_start = src_newline_ind[i]+1;

      while(total_parsed_cols < *num_cols_to_parse){
        // if its the column we want to parse
        if(ind[total_parsed_cols] == current_column) { //process
          dest_col_start[total_parsed_cols][i]=line_start+inline_pos;
          in_col_nonzeros = 0;
          while(source[line_start+inline_pos] != *field_separator){
            if(source[line_start+inline_pos] != 0) {
              in_col_nonzeros++;
            };
            inline_pos++;
          };
          //save the size of the column
          dest_col_size[total_parsed_cols][i]=in_col_nonzeros;
          inline_pos++;
          total_parsed_cols++;
        }
        // ignore the current column
        else{
          while(source[line_start+inline_pos] != *field_separator) {
            inline_pos++;
          };
          inline_pos++;
        };
        current_column++;
      }
    }
};

namespace OLAP{
  OLAP_Driver::~OLAP_Driver(){
    delete(scanner);
    scanner = nullptr;
    delete(parser);
    parser = nullptr;
  }

  void OLAP_Driver::parse( const char * const filename ){
    assert( filename != nullptr );
    std::ifstream in_file( filename );
    if( ! in_file.good() ){
      exit( EXIT_FAILURE );
    }
    parse_helper( in_file );
    return;
  }

  void OLAP_Driver::parse( std::istream &stream ){
    if( ! stream.good()  && stream.eof() ){
      return;
    }
    //else
    parse_helper( stream );
    return;
  }

  void OLAP_Driver::parse_helper( std::istream &stream ){
    delete(scanner);
    try{
      scanner = new OLAP::OLAP_Scanner( &stream );
    }
    catch( std::bad_alloc &ba ){
      std::cerr << "Failed to allocate scanner: (" <<
        ba.what() << "), exiting!!\n";
      exit( EXIT_FAILURE );
    }

    delete(parser);
    try{
      parser = new OLAP::OLAP_Parser( (*scanner) /* scanner */,
          (*this) /* driver */ );
    }
    catch( std::bad_alloc &ba ){
      std::cerr << "Failed to allocate parser: (" <<
        ba.what() << "), exiting!!\n";
      exit( EXIT_FAILURE );
    }
    const int accept( 0 );
    if( parser->parse() != accept ){
      std::cerr << "Parse failed!!\n";
    }
    return;
  }

  void OLAP_Driver::load_matrix_csc ( std::string  filename, int col_number, int max_col_size ){
    std::cout << "loading column " << col_number << " from file " << filename << std::endl;
    std::clock_t start1 = std::clock();
    int fd;
    fd = open ( filename.c_str(), O_RDONLY );
    if ( fd == -1 ){
      perror("open");
    }

    off_t file_size;
    file_size = lseek(fd, 0, SEEK_END);
    thrust::device_vector<char> dev(file_size);
    char* mapped_file;

    mapped_file = (char*)mmap (0, file_size, PROT_READ, MAP_SHARED, fd, 0);                  

    if (mapped_file == MAP_FAILED){                                                        
      perror ("mmap");                                                            
    }                                                                             

    if (close (fd) == -1){                                                       
      perror ("close");                                                           
    }                                                                             

    thrust::copy(mapped_file, mapped_file+file_size, dev.begin());                                     
    int line_count = std::count(dev.begin(), dev.end(), '\n');                        
    std::cout << "There are " << line_count << " total lines in a file with size " << file_size << std::endl;    

    // find out the position of every newline
    thrust::device_vector<int> dev_newline_pos(line_count+1); 
    thrust::copy_if(
        thrust::make_counting_iterator((unsigned int) 0 ), 
        thrust::make_counting_iterator((unsigned int) file_size), 
        dev.begin(), dev_newline_pos.begin()+1, 
        is_newline_break()
        ); 

    // field position based on column number
    thrust::device_vector<unsigned int> field_index(1);
    field_index[0]=col_number;

    // field separator
    thrust::device_vector<char> field_separator(1);
    field_separator[0] = '|';

    thrust::device_vector<int> dev_col_start1(line_count); 
    thrust::device_vector<int> dev_col_size1(line_count); 
    thrust::fill(dev_col_start1.begin(), dev_col_start1.end(), 0); 
    thrust::fill(dev_col_size1.begin(), dev_col_size1.end(), 0); 

    thrust::device_vector<int*> dest_col_start(1);
    thrust::device_vector<int*> dest_col_size(1);
    dest_col_start[0] = thrust::raw_pointer_cast(dev_col_start1.data());
    dest_col_size[0] = thrust::raw_pointer_cast(dev_col_size1.data());

    //fields max lengths 
    thrust::device_vector<unsigned int> dest_len(1); 
    dest_len[0] = max_col_size;

    //fields count
    thrust::device_vector<unsigned int> ind_cnt(1); 
    ind_cnt[0] = 1;

    thrust::counting_iterator<unsigned int> begin(0);

    parse_functor ff(
        (const char*) thrust::raw_pointer_cast(dev.data()),
        (int**) thrust::raw_pointer_cast(dest_col_start.data()), 
        (int**) thrust::raw_pointer_cast(dest_col_size.data()), 
        thrust::raw_pointer_cast(field_index.data()),
        thrust::raw_pointer_cast(ind_cnt.data()), 
        thrust::raw_pointer_cast(field_separator.data()), 
        thrust::raw_pointer_cast(dev_newline_pos.data()), 
        thrust::raw_pointer_cast(dest_len.data())
        );
    thrust::for_each(begin, begin + line_count, ff);
/*
    for (int pos = 0; pos < line_count ; pos ++  ){
      std::string col ( &(mapped_file[dev_col_start1[pos]]), dev_col_size1[pos] );
      std::cout << dev_col_start1[pos] << " " << dev_col_size1[pos] << " : " << col<<  std::endl;
    }
*/
    std::cout << "leaving load matrix " << std::endl;
  }

  std::ostream& OLAP_Driver::print( std::ostream &stream ){
    stream << red  << "Debug info: " << norm << "\n";
    stream << blue << "OLAP: "  << norm << "\n";

    return(stream);
  }
}


