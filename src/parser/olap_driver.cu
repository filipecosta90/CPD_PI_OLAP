#include "hip/hip_runtime.h"
#include <cctype>
#include <fstream>
#include <cassert>
#include <fcntl.h>
#include <sys/types.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/mman.h>

//GPU libs
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>

#include "olap_driver.hh"
#include "olap_parser.hh"
#include "olap_scanner.hh"

namespace OLAP{
  OLAP_Driver::~OLAP_Driver()
  {
    delete(scanner);
    scanner = nullptr;
    delete(parser);
    parser = nullptr;
  }

  void OLAP_Driver::parse( const char * const filename )
  {
    assert( filename != nullptr );
    std::ifstream in_file( filename );
    if( ! in_file.good() )
    {
      exit( EXIT_FAILURE );
    }
    parse_helper( in_file );
    return;
  }

  void OLAP_Driver::parse( std::istream &stream )
  {
    if( ! stream.good()  && stream.eof() )
    {
      return;
    }
    //else
    parse_helper( stream );
    return;
  }


  void OLAP_Driver::parse_helper( std::istream &stream )
  {

    delete(scanner);
    try
    {
      scanner = new OLAP::OLAP_Scanner( &stream );
    }
    catch( std::bad_alloc &ba )
    {
      std::cerr << "Failed to allocate scanner: (" <<
        ba.what() << "), exiting!!\n";
      exit( EXIT_FAILURE );
    }

    delete(parser);
    try
    {
      parser = new OLAP::OLAP_Parser( (*scanner) /* scanner */,
          (*this) /* driver */ );
    }
    catch( std::bad_alloc &ba )
    {
      std::cerr << "Failed to allocate parser: (" <<
        ba.what() << "), exiting!!\n";
      exit( EXIT_FAILURE );
    }
    const int accept( 0 );
    if( parser->parse() != accept )
    {
      std::cerr << "Parse failed!!\n";
    }
    return;
  }

  void OLAP_Driver::load_matrix_csc ( std::string  filename, int col_number ){
    std::cout << "loading column " << col_number << " from file " << filename << std::endl;
    std::clock_t start1 = std::clock();
    int fd;
    fd = open ( filename.c_str(), O_RDONLY );
    if ( fd == -1 ){
      perror("open");
    }

    off_t fsize;
    fsize = lseek(fd, 0, SEEK_END);
    thrust::device_vector<char> dev(fsize);
    char* p;

    p = (char*)mmap (0, fsize, PROT_READ, MAP_SHARED, fd, 0);                  

    if (p == MAP_FAILED) {                                                        
      perror ("mmap");                                                            
    }                                                                             

    if (close (fd) == -1) {                                                       
      perror ("close");                                                           
    }                                                                             

    thrust::copy(p, p+fsize, dev.begin());                                     
    std::cout << "going to count" << std::endl;
    int cnt = std::count(dev.begin(), dev.end(), '\n');                        
    std::cout << "There are " << cnt << " total lines in a file with size " << fsize << std::endl;    
    thrust::device_vector<int> dev_newline_pos(cnt+1); 
    /*thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int) fsize), dev.begin(), dev_newline_pos.begin()+1, is_newline_break()); 

     */
    thrust::device_vector<unsigned int> field_index(1);
    field_index[0]=col_number;
    thrust::device_vector<char> field_separator(1);
    field_separator[0] = '|';
    std::cout << "leaving load matrix " << std::endl;
  }


  std::ostream& OLAP_Driver::print( std::ostream &stream )
  {
    stream << red  << "Debug info: " << norm << "\n";
    stream << blue << "OLAP: "  << norm << "\n";

    return(stream);
  }
}


