#include "hip/hip_runtime.h"
#include <cctype>
#include <fstream>
#include <cassert>
#include <fcntl.h>
#include <sys/types.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/mman.h>

//GPU libs
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/fill.h>

//GLIB
#include "tbl_parser.cuh"

struct is_newline_break
{
  __host__ __device__
    bool operator()(const char x){
      return x == '\n';
    }
};

struct parse_functor
{
  const char *source;
  int **dest_col_start;
  int **dest_col_size;
  const unsigned int *ind;
  const unsigned int *num_cols_to_parse;
  const char *field_separator;
  const int *src_newline_ind;
  const unsigned int *dest_len;

  parse_functor(
      const char* _source, int** _dest_col_start, int** _dest_col_size, 
      const unsigned int* _ind, const unsigned int* _cnt, 
      const char* _separator,
      const int* _src_ind, const unsigned int* _dest_len
      ):
    source(_source), dest_col_start(_dest_col_start), dest_col_size(_dest_col_size), ind(_ind), num_cols_to_parse(_cnt),  field_separator(_separator), src_newline_ind(_src_ind), dest_len(_dest_len) {}

  template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
      unsigned int current_column = 0, total_parsed_cols = 0, inline_pos = 0, in_col_nonzeros, line_start;
      line_start = src_newline_ind[i]+1;

      while(total_parsed_cols < *num_cols_to_parse){
        // if its the column we want to parse
        if(ind[total_parsed_cols] == current_column) { //process
          //save the column start position in reference to the array start
          dest_col_start[total_parsed_cols][i]=line_start+inline_pos;
          in_col_nonzeros = 0;
          while(source[line_start+inline_pos] != *field_separator){
            if(source[line_start+inline_pos] != 0) {
              in_col_nonzeros++;
            };
            inline_pos++;
          };
          //save the size of the column
          dest_col_size[total_parsed_cols][i]=in_col_nonzeros;
          inline_pos++;
          total_parsed_cols++;
        }
        // ignore the current column
        else{
          while(source[line_start+inline_pos] != *field_separator) {
            inline_pos++;
          };
          inline_pos++;
        };
        current_column++;
      }
    }
};

struct quark_functor 
{ 
  const float a; 
  quark_functor(float _a) : a(_a) {} 
  __host__ __device__ 
    float operator()(const float& x, const float& y) const { 
      return a * x + y; 
    } 
};

namespace TBL{

  void TBL_Parser::load_matrix_csc ( 
      std::string filename, int col_number, int max_col_size, 
      int* n_nnz, int* n_rows, int* n_cols,
      float** __restrict__  A_csc_values,
      int** __restrict__  A_row_ind,
      int** __restrict__  A_col_ptr
      ){
    std::cout << "loading column " << col_number << " from file " << filename << std::endl;
    std::clock_t start1 = std::clock();
    int fd;
    fd = open ( filename.c_str(), O_RDONLY | O_NONBLOCK );
    if ( fd == -1 ){
      perror("open");
    }

    off_t file_size;
    file_size = lseek(fd, 0, SEEK_END);
    thrust::device_vector<char> dev(file_size);
    char* mapped_file;

    mapped_file = (char*)mmap (0, file_size, PROT_READ, MAP_SHARED, fd, 0);

    if (mapped_file == MAP_FAILED){
      perror ("mmap");
    }

    if (close (fd) == -1){
      perror ("close");
    }

    thrust::copy(mapped_file, mapped_file+file_size, dev.begin());
    int line_count = std::count(dev.begin(), dev.end(), '\n');
    std::cout << "There are " << line_count << " total lines in a file with size " << file_size << std::endl;

    // find out the position of every newline
    thrust::device_vector<int> dev_newline_pos(line_count+1); 
    thrust::copy_if(
        thrust::make_counting_iterator((unsigned int) 0 ), 
        thrust::make_counting_iterator((unsigned int) file_size), 
        dev.begin(), dev_newline_pos.begin()+1, 
        is_newline_break()
        ); 

    // field position based on column number
    thrust::device_vector<unsigned int> field_index(1);
    field_index[0]=col_number;

    // field separator
    thrust::device_vector<char> field_separator(1);
    field_separator[0] = '|';

    thrust::device_vector<int> dev_col_start1(line_count); 
    thrust::device_vector<int> dev_col_size1(line_count); 
    thrust::fill(dev_col_start1.begin(), dev_col_start1.end(), 0); 
    thrust::fill(dev_col_size1.begin(), dev_col_size1.end(), 0); 

    thrust::device_vector<int*> dest_col_start(1);
    thrust::device_vector<int*> dest_col_size(1);
    dest_col_start[0] = thrust::raw_pointer_cast(dev_col_start1.data());
    dest_col_size[0] = thrust::raw_pointer_cast(dev_col_size1.data());

    //fields max lengths 
    thrust::device_vector<unsigned int> dest_len(1); 
    dest_len[0] = max_col_size;

    //fields count
    thrust::device_vector<unsigned int> ind_cnt(1); 
    ind_cnt[0] = 1;

    thrust::counting_iterator<unsigned int> begin(0);

    parse_functor ff(
        (const char*) thrust::raw_pointer_cast(dev.data()),
        (int**) thrust::raw_pointer_cast(dest_col_start.data()), 
        (int**) thrust::raw_pointer_cast(dest_col_size.data()), 
        thrust::raw_pointer_cast(field_index.data()),
        thrust::raw_pointer_cast(ind_cnt.data()), 
        thrust::raw_pointer_cast(field_separator.data()), 
        thrust::raw_pointer_cast(dev_newline_pos.data()), 
        thrust::raw_pointer_cast(dest_len.data())
        );

    thrust::for_each(begin, begin + line_count, ff);

    thrust::device_vector<int> aux_csc_col_ptr (line_count+1);
    thrust::host_vector<int> aux_csc_row_ind (line_count);
    thrust::device_vector<float> aux_csc_values (line_count);

    // initialize aux_csc_col_ptr to 0,1,2,3, (line_count+1) .... thrust::sequence(aux_csc_col_ptr.begin(), aux_csc_col_ptr.end());
    thrust::sequence(aux_csc_col_ptr.begin(), aux_csc_col_ptr.end());

    // fill aux_csc_values with ones
    thrust::fill(aux_csc_values.begin(), aux_csc_values.end(), 1.0f);

    for (int pos = 0; pos < line_count ; pos ++  ){
      const std::string element ( &(mapped_file[dev_col_start1[pos]]), dev_col_size1[pos] );
      const int row_of_element = 0;//= engine->get_row_from_string( element ); 
      aux_csc_row_ind[pos] = row_of_element;
      std::cout << dev_col_start1[pos] << " " << dev_col_size1[pos] << " : " << element << " | " << row_of_element <<  std::endl;
    }

    thrust::device_vector<int> dev_aux_csc_row_ind = aux_csc_row_ind;
    float current_major_row  = thrust::reduce(dev_aux_csc_row_ind.begin(), dev_aux_csc_row_ind.end(), (int) 0, thrust::maximum<float>());
    current_major_row++;
    *n_rows = current_major_row; 
    *n_cols = line_count;
    *n_nnz = line_count; 

    // extract raw pointer from the device vector of aux_csc_values.data
    *A_csc_values = thrust::raw_pointer_cast(aux_csc_values.data());

    // extract raw pointer from the device vector of aux_csc_col_ptr.data
    // JA  points to column starts in A 
    *A_col_ptr = thrust::raw_pointer_cast(aux_csc_col_ptr.data());

    // extract raw pointer from the device vector of aux_csc_row_ind.data
    // IA splits the array A into rows
    *A_row_ind = thrust::raw_pointer_cast(aux_csc_row_ind.data());

  }

}

